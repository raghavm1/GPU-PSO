#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include "common.h"
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

// cuda constant memory
__constant__ double w_d;
__constant__ double c1_d;
__constant__ double c2_d;
__constant__ double max_pos_d;
__constant__ double min_pos_d;
__constant__ double max_v_d;
__constant__ int max_iter_d;
__constant__ int particle_cnt_d;
__constant__ int tile_size;
__constant__ int tile_size2;

__host__ __device__ double fit(double x)
{
    return fabs(8000.0 + x * (-10000.0 + x * (-0.8 + x)));
}

__global__ void updateParticles(double *position_d, double *velocity_d, double *fitness_d,
                                double *pbest_pos_d, double *pbest_fit_d,
                                double *best_fitness_buf, double *best_positions_buf, int dim_d, double *gbest_position_d, double *gbest_fitness_d)
{
    int particle_idx = blockIdx.x;
    int dim_idx = threadIdx.x;
    extern __shared__ double shared_mem[];
    double *particle_pos = &shared_mem[0];
    double *particle_vel = &shared_mem[dim_d];
    double *particle_pbest = &shared_mem[2 * dim_d];
    int idx = particle_idx * dim_d + dim_idx;

    if (dim_idx < dim_d)
    {

        particle_pos[dim_idx] = position_d[idx];
        particle_vel[dim_idx] = velocity_d[idx];
        particle_pbest[dim_idx] = pbest_pos_d[idx];
    }
    __syncthreads();

    if (dim_idx < dim_d)
    {

        hiprandState state;
        hiprand_init((unsigned long long)clock() + idx, 0, 0, &state);

        double r1 = hiprand_uniform_double(&state);
        double r2 = hiprand_uniform_double(&state);
        double v = particle_vel[dim_idx];

        double pos = particle_pos[dim_idx];

        double pbest = double(particle_pbest[dim_idx]);

        double gbest_pos = double(gbest_position_d[dim_idx]);

        v = w_d * v +
            c1_d * r1 * (pbest - pos) +
            c2_d * r2 * (gbest_pos - pos);

        v = max(min(v, max_v_d), -max_v_d);

        pos = pos + v;

        pos = max(min(pos, max_pos_d), min_pos_d);

        particle_pos[dim_idx] = pos;
        particle_vel[dim_idx] = v;

        position_d[idx] = pos;
        velocity_d[idx] = v;
    }
    __syncthreads();

    if (threadIdx.x == 0 && particle_idx < particle_cnt_d)
    {
        double fitness = 0.0;

        for (int i = 0; i < dim_d; i++)
        {
            fitness += fit(particle_pos[i]);
        }

        fitness_d[particle_idx] = fitness;

        if (fitness > pbest_fit_d[particle_idx])
        {
            pbest_fit_d[particle_idx] = fitness;

            // Store all dimensions of this particle's position
            for (int i = 0; i < dim_d; i++)
            {
                pbest_pos_d[particle_idx * dim_d + i] = particle_pos[i];
            }
        }

        // Store fitness and all position dimensions for global best finding
        best_fitness_buf[particle_idx] = fitness;
        for (int i = 0; i < dim_d; i++)
        {
            best_positions_buf[particle_idx * dim_d + i] = particle_pos[i];
        }
    }
}

__global__ void findGlobalBest(double *best_fitness_buf, double *best_positions_buf, int particle_count, int dim_d, double *gbest_fitness_d, double *gbest_position_d)
{
    extern __shared__ double shared_data[];
    double *s_fitness = &shared_data[0];
    double *s_indices = &shared_data[blockDim.x];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid < particle_count)
    {
        s_fitness[tid] = best_fitness_buf[gid];
        s_indices[tid] = gid;
    }
    else
    {
        s_fitness[tid] = -INFINITY;
        s_indices[tid] = -1;
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride && gid < particle_count)
        {
            if (s_fitness[tid] < s_fitness[tid + stride])
            {
                s_fitness[tid] = s_fitness[tid + stride];
                s_indices[tid] = s_indices[tid + stride];
            }
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        if (s_fitness[0] > gbest_fitness_d[0])
        {
            int best_idx = (int)s_indices[0];
            if (best_idx >= 0)
            {
                gbest_fitness_d[0] = s_fitness[0];
                for (int d = 0; d < dim_d; d++)
                {
                    gbest_position_d[d] = best_positions_buf[best_idx * dim_d + d];
                }
            }
        }
    }
}

void ParticleInitCoal(particle_Coal *p, int dimensions)
{
    const double pos_range = max_pos - min_pos;
    srand((unsigned)time(NULL));

    p->position = (double *)malloc(sizeof(double) * particle_cnt * dimensions);
    p->velocity = (double *)malloc(sizeof(double) * particle_cnt * dimensions);
    p->fitness = (double *)malloc(sizeof(double) * particle_cnt);
    p->pbest_pos = (double *)malloc(sizeof(double) * particle_cnt * dimensions);
    p->pbest_fit = (double *)malloc(sizeof(double) * particle_cnt);

    for (int i = 0; i < particle_cnt; i++)
    {
        double fitness = 0.0;

        for (int d = 0; d < dimensions; d++)
        {
            int idx = i * dimensions + d;
            p->position[idx] = double(RND() * pos_range + min_pos);
            p->velocity[idx] = RND() * max_v;
            p->pbest_pos[idx] = double(p->position[idx]);

            fitness += fit(p->position[idx]);
        }

        p->fitness[i] = fitness;
        p->pbest_fit[i] = fitness;

        if (fitness > gbest->g_fitness[0])
        {
            gbest->g_fitness[0] = fitness;
            for (int d = 0; d < dimensions; d++)
            {
                gbest->position[d] = p->position[i * dimensions + d];
            }
        }
    }
}

int main(int argc, char **argv)
{
    arguments args = {10000, 1024, 1024, 4, 3, 4};
    int retError = pargeArgs(&args, argc, argv);
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    float exe_time;
    clock_t begin_app = clock();
    clock_t begin_init = begin_app;
    particle_Coal *p; // p : 粒子群
    p = (particle_Coal *)malloc(sizeof(particle_Coal));
    double *position_d;
    double *velocity_d;
    double *fitness_d;
    double *pbest_pos_d;
    double *pbest_fit_d;
    double *aux, *aux_pos;
    double *gbest_position_d;
    double *gbest_fitness_d;
    int *lock_d; // block level lock for gbest
    int block_size = min(1024, args.blocks_per_grid);

    int dim_d = args.dimensions;
    min_pos = -100.0, max_pos = +100.0;
    w = 1, c1 = 2.0, c2 = 2.0;
    particle_cnt = args.particle_cnt;
    max_v = (max_pos - min_pos) * 1.0;

    HANDLE_ERROR(hipMalloc((void **)&gbest_position_d, sizeof(double) * args.dimensions));
    HANDLE_ERROR(hipMalloc((void **)&gbest_fitness_d, sizeof(double) * 1));
    gbest = (particle *)malloc(sizeof(particle));
    initialize_gbest(gbest, args.dimensions);
    ParticleInitCoal(p, args.dimensions);
    int dimensions = args.dimensions;
    HANDLE_ERROR(hipMalloc((void **)&position_d, sizeof(double) * particle_cnt * dimensions));
    HANDLE_ERROR(hipMalloc((void **)&velocity_d, sizeof(double) * particle_cnt * dimensions));
    HANDLE_ERROR(hipMalloc((void **)&fitness_d, sizeof(double) * particle_cnt));
    HANDLE_ERROR(hipMalloc((void **)&pbest_pos_d, sizeof(double) * dimensions * particle_cnt));
    HANDLE_ERROR(hipMalloc((void **)&pbest_fit_d, sizeof(double) * particle_cnt));
    HANDLE_ERROR(hipMalloc((void **)&lock_d, sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&aux, sizeof(double) * args.blocks_per_grid));
    HANDLE_ERROR(hipMalloc((void **)&aux_pos, sizeof(double) * args.blocks_per_grid));
    HANDLE_ERROR(hipMemcpy(position_d, p->position, sizeof(double) * particle_cnt * dimensions, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(velocity_d, p->velocity, sizeof(double) * particle_cnt * dimensions, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(fitness_d, p->fitness, sizeof(double) * particle_cnt, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(pbest_pos_d, p->pbest_pos, sizeof(double) * dimensions * particle_cnt, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(pbest_fit_d, p->pbest_fit, sizeof(double) * particle_cnt, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(gbest_position_d, gbest->position, sizeof(double) * args.dimensions, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(gbest_fitness_d, gbest->g_fitness, sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(w_d), &w, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c1_d), &c1, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c2_d), &c2, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(max_pos_d), &max_pos, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(min_pos_d), &min_pos, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(max_v_d), &max_v, sizeof(double)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(max_iter_d), &args.max_iter, sizeof(int)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(particle_cnt_d), &args.particle_cnt, sizeof(int)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(tile_size), &args.threads_per_block, sizeof(int)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(tile_size2), &block_size, sizeof(int)));
    HANDLE_ERROR(hipMemset(lock_d, 0, sizeof(int)));
    clock_t end_init = clock();
    HANDLE_ERROR(hipEventRecord(start));

    double *best_fitness_buf_d;
    double *best_positions_buf_d;
    HANDLE_ERROR(hipMalloc((void **)&best_fitness_buf_d,
                            sizeof(double) * particle_cnt));
    HANDLE_ERROR(hipMalloc((void **)&best_positions_buf_d,
                            sizeof(double) * particle_cnt * dimensions));

    size_t particle_shared_mem = 3 * dimensions * sizeof(double);

    size_t reduction_shared_mem = 2 * block_size * sizeof(double);

    for (unsigned int i = 0; i < args.max_iter; i++)
    {

        updateParticles<<<particle_cnt, dimensions, particle_shared_mem>>>(
            position_d, velocity_d, fitness_d,
            pbest_pos_d, pbest_fit_d,
            best_fitness_buf_d, best_positions_buf_d, dim_d, gbest_position_d, gbest_fitness_d);

        // Find global best
        findGlobalBest<<<1, block_size, reduction_shared_mem>>>(
            best_fitness_buf_d, best_positions_buf_d, particle_cnt, dim_d, gbest_fitness_d, gbest_position_d);
    }
    HANDLE_ERROR(hipEventRecord(stop));

    HANDLE_ERROR(hipMemcpy(p->position, position_d, sizeof(double) * particle_cnt * dimensions, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(p->velocity, velocity_d, sizeof(double) * particle_cnt * dimensions, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(p->fitness, fitness_d, sizeof(double) * particle_cnt, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(p->pbest_pos, pbest_pos_d, sizeof(double) * dimensions * particle_cnt, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(p->pbest_fit, pbest_fit_d, sizeof(double) * particle_cnt, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipMemcpy(gbest->g_fitness, gbest_fitness_d, sizeof(double), hipMemcpyDeviceToHost));

    clock_t end_exe = clock();
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&exe_time, start, stop));

    free(p);
    hipFree(position_d);
    hipFree(velocity_d);
    hipFree(fitness_d);
    hipFree(pbest_pos_d);
    hipFree(pbest_fit_d);
    hipFree(lock_d);

    printf("best result: %lf\n", gbest->g_fitness[0]);
    printf("[Initial   time]: %lf (sec)\n", (double)(end_init - begin_init) / CLOCKS_PER_SEC);
    printf("[Cuda Exec time]: %f (sec)\n", exe_time / 1000);
    printf("[Elapsed   time]: %lf (sec)\n", (double)(clock() - begin_app) / CLOCKS_PER_SEC);
    hipFree(best_fitness_buf_d);
    hipFree(best_positions_buf_d);
    free_gbest();

    return 0;
}
